#include "airfoil_cuda_matrices.h"

void internal_fluxes_matrices(hipblasHandle_t handle, const int numCells,
                              const double *F_d, const double *G_d,
                              double *dFdr_d, double *dFds_d, double *dGdr_d,
                              double *dGds_d) {
  double *ones_d;
  hipMalloc((void**)&ones_d, 15 * sizeof(double));
  hipMemcpy(ones_d, ones, 15 * sizeof(double), hipMemcpyHostToDevice);

  double *r_d;
  hipMalloc((void**)&r_d, 15 * sizeof(double));
  hipMemcpy(r_d, r, 15 * sizeof(double), hipMemcpyHostToDevice);

  double *s_d;
  hipMalloc((void**)&s_d, 15 * sizeof(double));
  hipMemcpy(s_d, s, 15 * sizeof(double), hipMemcpyHostToDevice);

  double *temp_d;
  hipMalloc((void**)&temp_d, numCells * 15 * sizeof(double));

  double *Dr_d;
  hipMalloc((void**)&Dr_d, 15 * 15 * sizeof(double));
  hipMemcpy(Dr_d, Dr, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);

  double *Ds_d;
  hipMalloc((void**)&Ds_d, 15 * 15 * sizeof(double));
  hipMemcpy(Ds_d, Ds, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  for(int c = 0; c < numCells; c++) {
    const double *F = F_d + c * 4 * 15;
    const double *G = G_d + c * 4 * 15;
    double *dFdr = dFdr_d + c * 4 * 15;
    double *dFds = dFds_d + c * 4 * 15;
    double *dGdr = dGdr_d + c * 4 * 15;
    double *dGds = dGds_d + c * 4 * 15;

    for(int i = 0; i < 4; i++) {
      cblas_dgemv(CblasRowMajor, CblasNoTrans, 15, 15, 1.0, Drw, 15, &F[i], 4, 0.0, dFdr, 1);
      cblas_dgemv(CblasRowMajor, CblasNoTrans, 15, 15, 1.0, Dsw, 15, &F[i], 4, 0.0, dFds, 1);
      cblas_dgemv(CblasRowMajor, CblasNoTrans, 15, 15, 1.0, Drw, 15, &G[i], 4, 0.0, dGdr, 1);
      cblas_dgemv(CblasRowMajor, CblasNoTrans, 15, 15, 1.0, Dsw, 15, &G[i], 4, 0.0, dGds, 1);
    }
  }

  hipFree(ones_d);
  hipFree(r_d);
  hipFree(temp_d);
  hipFree(Dr_d);
  hipFree(Ds_d);
}
