#include "airfoil_cuda_matrices.h"

void internal_fluxes_matrices(hipblasHandle_t handle, const int numCells,
                              const double *F_d, const double *G_d,
                              double *dFdr_d, double *dFds_d, double *dGdr_d,
                              double *dGds_d) {
  double *Drw_d;
  hipMalloc((void**)&Drw_d, 15 * 15 * sizeof(double));
  hipMemcpy(Drw_d, Drw, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);

  double *Dsw_d;
  hipMalloc((void**)&Dsw_d, 15 * 15 * sizeof(double));
  hipMemcpy(Dsw_d, Dsw, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  hipStream_t streams[NUMBER_OF_MATRIX_STREAMS];
  for(int i = 0; i < NUMBER_OF_MATRIX_STREAMS; i++) {
    hipStreamCreate(&streams[i]);
  }
  for(int c = 0; c < numCells; c++) {
    hipblasSetStream(handle, streams[c % NUMBER_OF_MATRIX_STREAMS]);
    const double *F = F_d + c * 4 * 15;
    const double *G = G_d + c * 4 * 15;
    double *dFdr = dFdr_d + c * 4 * 15;
    double *dFds = dFds_d + c * 4 * 15;
    double *dGdr = dGdr_d + c * 4 * 15;
    double *dGds = dGds_d + c * 4 * 15;

    for(int i = 0; i < 4; i++) {
      double alpha = 1.0;
      double beta = 0.0;
      // HIPBLAS_OP_T because cublas is column major but constants are stored row major
      hipblasDgemv(handle, HIPBLAS_OP_T, 15, 15, &alpha, Drw_d, 15, F + i, 4, &beta, dFdr + i, 4);
      hipblasDgemv(handle, HIPBLAS_OP_T, 15, 15, &alpha, Dsw_d, 15, F + i, 4, &beta, dFds + i, 4);
      hipblasDgemv(handle, HIPBLAS_OP_T, 15, 15, &alpha, Drw_d, 15, G + i, 4, &beta, dGdr + i, 4);
      hipblasDgemv(handle, HIPBLAS_OP_T, 15, 15, &alpha, Dsw_d, 15, G + i, 4, &beta, dGds + i, 4);
    }
  }

  for(int i = 0; i < NUMBER_OF_MATRIX_STREAMS; i++) {
    hipStreamDestroy(streams[i]);
  }

  hipFree(Drw_d);
  hipFree(Dsw_d);
  hipDeviceSynchronize();
}
