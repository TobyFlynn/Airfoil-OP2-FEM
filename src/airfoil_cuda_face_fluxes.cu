#include "airfoil_cuda_matrices.h"

void face_fluxes_matrices(hipblasHandle_t handle, const int numCells,
                          const double *flux0_d, const double *flux1_d,
                          const double *flux2_d, const double *flux3_d,
                          double *qRHS0_d, double *qRHS1_d, double *qRHS2_d,
                          double *qRHS3_d) {
  double *LIFT_d;
  hipMalloc((void**)&LIFT_d, 15 * 15 * sizeof(double));
  hipMemcpy(LIFT_d, LIFT, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);

  double alpha = -1.0;
  double beta = 1.0;

  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha, LIFT_d, 15, flux0_d, 15, &beta, qRHS0_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha, LIFT_d, 15, flux1_d, 15, &beta, qRHS1_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha, LIFT_d, 15, flux2_d, 15, &beta, qRHS2_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha, LIFT_d, 15, flux3_d, 15, &beta, qRHS3_d, 15);

  hipDeviceSynchronize();
  hipFree(LIFT_d);
}
