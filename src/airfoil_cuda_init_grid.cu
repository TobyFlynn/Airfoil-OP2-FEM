#include "airfoil_cuda_matrices.h"

void init_grid_matrices(hipblasHandle_t handle, const int numCells,
                        const double *node_coords, const int *cell2nodes,
                        double *x_d, double *y_d, double *xr_d, double *xs_d,
                        double *yr_d, double *ys_d) {
  hipDeviceSynchronize();
  double *ones_d;
  hipMalloc((void**)&ones_d, 15 * sizeof(double));
  hipMemcpy(ones_d, ones, 15 * sizeof(double), hipMemcpyHostToDevice);

  double *r_d;
  hipMalloc((void**)&r_d, 15 * sizeof(double));
  hipMemcpy(r_d, r, 15 * sizeof(double), hipMemcpyHostToDevice);

  double *s_d;
  hipMalloc((void**)&s_d, 15 * sizeof(double));
  hipMemcpy(s_d, s, 15 * sizeof(double), hipMemcpyHostToDevice);

  double *temp_d;
  hipMalloc((void**)&temp_d, numCells * 15 * sizeof(double));

  double *Dr_d;
  hipMalloc((void**)&Dr_d, 15 * 15 * sizeof(double));
  hipMemcpy(Dr_d, Dr, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);

  double *Ds_d;
  hipMalloc((void**)&Ds_d, 15 * 15 * sizeof(double));
  hipMemcpy(Ds_d, Ds, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  for(int c = 0; c < numCells; c++) {
    // Get nodes for this cell (on host)
    const double *n0 = &node_coords[2 * cell2nodes[3 * c]];
    const double *n1 = &node_coords[2 * cell2nodes[3 * c + 1]];
    const double *n2 = &node_coords[2 * cell2nodes[3 * c + 2]];

    double *temp = temp_d + c * 15;
    double *x = x_d + c * 15;
    double *y = y_d + c * 15;
    double *xr = xr_d + c * 15;
    double *xs = xs_d + c * 15;
    double *yr = yr_d + c * 15;
    double *ys = ys_d + c * 15;

    double alpha = 1.0;
    hipblasDcopy(handle, 15, ones_d, 1, x, 1);
    hipblasDaxpy(handle, 15, &alpha, r_d, 1, x, 1);
    alpha = 0.5 * n1[0];
    hipblasDscal(handle, 15, &alpha, x, 1);
    hipblasDcopy(handle, 15, ones_d, 1, temp, 1);
    alpha = 1.0;
    hipblasDaxpy(handle, 15, &alpha, s_d, 1, temp, 1);
    alpha = 0.5 * n2[0];
    hipblasDaxpy(handle, 15, &alpha, temp, 1, x, 1);
    hipblasDcopy(handle, 15, s_d, 1, temp, 1);
    alpha = 1.0;
    hipblasDaxpy(handle, 15, &alpha, r_d, 1, temp, 1);
    alpha = -0.5 * n0[0];
    hipblasDaxpy(handle, 15, &alpha, temp, 1, x, 1);

    hipblasDcopy(handle, 15, ones_d, 1, y, 1);
    alpha = 1.0;
    hipblasDaxpy(handle, 15, &alpha, r_d, 1, y, 1);
    alpha = 0.5 * n1[1];
    hipblasDscal(handle, 15, &alpha, y, 1);
    hipblasDcopy(handle, 15, ones_d, 1, temp, 1);
    alpha = 1.0;
    hipblasDaxpy(handle, 15, &alpha, s_d, 1, temp, 1);
    alpha = 0.5 * n2[1];
    hipblasDaxpy(handle, 15, &alpha, temp, 1, y, 1);
    hipblasDcopy(handle, 15, s_d, 1, temp, 1);
    alpha = 1.0;
    hipblasDaxpy(handle, 15, &alpha, r_d, 1, temp, 1);
    alpha = -0.5 * n0[1];
    hipblasDaxpy(handle, 15, &alpha, temp, 1, y, 1);

    // HIPBLAS_OP_T because cublas is column major but constants are stored row major
    // xr = Dr * x
    alpha = 1.0;
    double beta = 0.0;
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 15, &alpha, Dr_d, 15, x, 1, &beta, xr, 1);
    // xs = Ds * x
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 15, &alpha, Ds_d, 15, x, 1, &beta, xs, 1);
    // yr = Dr * y
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 15, &alpha, Dr_d, 15, y, 1, &beta, yr, 1);
    // ys = Ds * y
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 15, &alpha, Ds_d, 15, y, 1, &beta, ys, 1);
  }

  hipFree(ones_d);
  hipFree(r_d);
  hipFree(temp_d);
  hipFree(Dr_d);
  hipFree(Ds_d);
  hipDeviceSynchronize();
}
